#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <thrust/complex.h>

struct complex_abs_functor
{
    __host__ thrust::complex<float> operator()(const thrust::complex<float> z) {
        // return thrust::asinh(z);
        // return thrust::atanh(z);
        return thrust::acosh(z);
    }
};

int main(void) {
    thrust::host_vector<thrust::complex<float>> H(4);
    H[0] = 0.95;
    H[1] = 20.23;
    H[2] = 38.23;
    H[3] = 46.23;
    std::cout << "Size: " << H.size() << std::endl;
    thrust::host_vector<thrust::complex<float>> OUT(4);
    thrust::transform(H.begin(), H.end(), OUT.begin(), complex_abs_functor());
    std::cout << OUT[0] << ", " << OUT[1] << ", " << OUT[2] << ", " << OUT[3] << std::endl;
}
/*
int main(void)
{
  // generate random data serially
  thrust::host_vector<int> h_vec(100);
  std::generate(h_vec.begin(), h_vec.end(), rand);

  // transfer to device and compute sum
  thrust::device_vector<int> d_vec = h_vec;
  int x = thrust::reduce(d_vec.begin(), d_vec.end(), 0, thrust::plus<int>());
  std::cout << "X: " << x << std::endl;
  return 0;
}*/
